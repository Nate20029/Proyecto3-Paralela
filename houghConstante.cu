#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : make
 ============================================================================
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int tolerance = 1;
const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;
__constant__ float d_Cos[degreeBins];
__constant__ float d_Sin[degreeBins];

//*****************************************************************
// The CPU function returns a pointer to the accummulator
void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
  *acc = new int[rBins * degreeBins];                // el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
  memset(*acc, 0, sizeof(int) * rBins * degreeBins); // init en ceros
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++)   // por cada pixel
    for (int j = 0; j < h; j++) //...
    {
      int idx = j * w + i;
      if (pic[idx] > 0) // si pasa thresh, entonces lo marca
      {
        int xCoord = i - xCent;
        int yCoord = yCent - j;                       // y-coord has to be reversed
        float theta = 0;                              // actual angle
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) // add 1 to all lines in that pixel
        {
          float r = xCoord * cos(theta) + yCoord * sin(theta);
          int rIdx = (r + rMax) / rScale;
          (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
          theta += radInc;
        }
      }
    }
}

//*****************************************************************
// TODO usar memoria constante para la tabla de senos y cosenos
// inicializarlo en main y pasarlo al device
//__constant__ float d_Cos[degreeBins];
//__constant__ float d_Sin[degreeBins];

//*****************************************************************
// TODO Kernel memoria compartida
// __global__ void GPU_HoughTranShared(...)
// {
//   //TODO
// }
// TODO Kernel memoria Constante
// __global__ void GPU_HoughTranConst(...)
// {
//   //TODO
// }

// GPU kernel. One thread per image pixel is spawned.
// The accummulator memory needs to be allocated by the host in global memory
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale)
{
  // TODO calcular: int gloID = ?
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID >= w * h)
    return;

  int xCent = w / 2;
  int yCent = h / 2;

  // TODO explicar bien bien esta parte. Dibujar un rectangulo a modo de imagen sirve para visualizarlo mejor
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  // TODO eventualmente usar memoria compartida para el acumulador

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      // TODO utilizar memoria constante para senos y cosenos
      // float r = xCoord * cos(tIdx) + yCoord * sin(tIdx); //probar con esto para ver diferencia en tiempo
      float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
      int rIdx = (r + rMax) / rScale;
      // debemos usar atomic, pero que race condition hay si somos un thread por pixel? explique
      atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
    }
  }

  // TODO eventualmente cuando se tenga memoria compartida, copiar del local al global
  // utilizar operaciones atomicas para seguridad
  // faltara sincronizar los hilos del bloque en algunos lados
}

//*****************************************************************
int main(int argc, char **argv)
{
  if (argc != 2)
  {
    printf("Usage: %s <image.pgm>\n", argv[0]);
    return -1;
  }

  int i;

  PGMImage inImg(argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;


  float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
  float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
  float rad = 0;

  hipMalloc((void **)&d_Cos, sizeof(float) * degreeBins);
  hipMalloc((void **)&d_Sin, sizeof(float) * degreeBins);

  // CPU calculation
  CPU_HoughTran(inImg.pixels, w, h, &cpuht);


  for (i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos(rad);
    pcSin[i] = sin(rad);
    rad += radInc;
  }

  float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  hipMemcpyToSymbol(HIP_SYMBOL(d_Cos), pcCos, sizeof(float) * degreeBins);
  hipMemcpyToSymbol(HIP_SYMBOL(d_Sin), pcSin, sizeof(float) * degreeBins);

  unsigned char *d_in;
  int *d_hough;
  int *h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

  hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
  hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
  hipMemcpy(d_in, inImg.pixels, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

  // Define CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start, NULL);

  // Launch the kernel
  int blockNum = ceil((float)w * h / 256.0);
  GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale);

  // Record the stop event
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);

  // Calculate and print the elapsed time
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Hough Constante Transform tomo %f milisegundos\n", milliseconds);

  // Copy results back to host
  hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  // Compare CPU and GPU results
  for (i = 0; i < degreeBins * rBins; i++)
  {
    if (abs(cpuht[i] != h_hough[i]) > tolerance)
      printf("Mismatch at index %d: CPU=%d, GPU=%d\n", i, cpuht[i], h_hough[i]);
  }

  // Free dynamically allocated memory
  hipFree(d_in);
  hipFree(d_hough);
  free(pcCos);
  free(pcSin);
  free(h_hough);
  delete[] cpuht;

  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  printf("Done!\n");

  return 0;
}
