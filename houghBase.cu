#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include "common/pgm.h"

const int tolerance = 1;
const int degreeInc = 2;
const int degreeBins = 180 / degreeInc;
const int rBins = 100;
const float radInc = degreeInc * M_PI / 180;

void CPU_HoughTran(unsigned char *pic, int w, int h, int **acc)
{
  float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
  *acc = new int[rBins * degreeBins];
  memset(*acc, 0, sizeof(int) * rBins * degreeBins);
  int xCent = w / 2;
  int yCent = h / 2;
  float rScale = 2 * rMax / rBins;

  for (int i = 0; i < w; i++)
    for (int j = 0; j < h; j++)
    {
      int idx = j * w + i;
      if (pic[idx] > 0)
      {
        int xCoord = i - xCent;
        int yCoord = yCent - j;
        float theta = 0;
        for (int tIdx = 0; tIdx < degreeBins; tIdx++)
        {
          float r = xCoord * cos(theta) + yCoord * sin(theta);
          int rIdx = (r + rMax) / rScale;
          (*acc)[rIdx * degreeBins + tIdx]++;
          theta += radInc;
        }
      }
    }
}

__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, float rMax, float rScale, float *d_Cos, float *d_Sin)
{
  int gloID = blockIdx.x * blockDim.x + threadIdx.x;
  if (gloID >= w * h)
    return;

  int xCent = w / 2;
  int yCent = h / 2;
  int xCoord = gloID % w - xCent;
  int yCoord = yCent - gloID / w;

  if (pic[gloID] > 0)
  {
    for (int tIdx = 0; tIdx < degreeBins; tIdx++)
    {
      float r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
      int rIdx = (r + rMax) / rScale;
      atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
    }
  }
}

int main(int argc, char **argv)
{
  if (argc != 2)
  {
    printf("Usage: %s <image.pgm>\n", argv[0]);
    return -1;
  }

  PGMImage inImg(argv[1]);

  int *cpuht;
  int w = inImg.x_dim;
  int h = inImg.y_dim;

  float *d_Cos;
  float *d_Sin;

  hipMalloc((void **)&d_Cos, sizeof(float) * degreeBins);
  hipMalloc((void **)&d_Sin, sizeof(float) * degreeBins);

  CPU_HoughTran(inImg.pixels, w, h, &cpuht);

  float *pcCos = (float *)malloc(sizeof(float) * degreeBins);
  float *pcSin = (float *)malloc(sizeof(float) * degreeBins);
  float rad = 0;
  for (int i = 0; i < degreeBins; i++)
  {
    pcCos[i] = cos(rad);
    pcSin[i] = sin(rad);
    rad += radInc;
  }

  float rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
  float rScale = 2 * rMax / rBins;

  hipMemcpy(d_Cos, pcCos, sizeof(float) * degreeBins, hipMemcpyHostToDevice);
  hipMemcpy(d_Sin, pcSin, sizeof(float) * degreeBins, hipMemcpyHostToDevice);

  unsigned char *d_in;
  int *d_hough;
  int *h_hough = (int *)malloc(degreeBins * rBins * sizeof(int));

  hipMalloc((void **)&d_in, sizeof(unsigned char) * w * h);
  hipMalloc((void **)&d_hough, sizeof(int) * degreeBins * rBins);
  hipMemcpy(d_in, inImg.pixels, sizeof(unsigned char) * w * h, hipMemcpyHostToDevice);
  hipMemset(d_hough, 0, sizeof(int) * degreeBins * rBins);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, NULL);

  int blockNum = ceil((float)w * h / 256.0);
  GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Hough Transform took %f milliseconds\n", milliseconds);

  hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

  for (int i = 0; i < degreeBins * rBins; i++)
  {
    if (abs(cpuht[i] != h_hough[i]) > tolerance)
      printf("Mismatch at index %d: CPU=%d, GPU=%d\n", i, cpuht[i], h_hough[i]);
  }

  hipFree(d_Cos);
  hipFree(d_Sin);
  hipFree(d_in);
  hipFree(d_hough);
  free(pcCos);
  free(pcSin);
  free(h_hough);
  delete[] cpuht;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  printf("Done!\n");

  return 0;
}
